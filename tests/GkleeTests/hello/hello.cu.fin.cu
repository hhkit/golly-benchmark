// taken from http://computer-graphics.se/hello-world-for-cuda.html
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(char *a, int *b) 
{
  a[threadIdx.x] += b[threadIdx.x];
}
 

