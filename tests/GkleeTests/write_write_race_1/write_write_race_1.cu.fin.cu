
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void device_global(unsigned int *input_array, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;
  input_array[0] = my_index;
  }


