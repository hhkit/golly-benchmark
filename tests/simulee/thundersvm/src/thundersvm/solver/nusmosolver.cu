#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-10-25.
//
#include <thundersvm/solver/nusmosolver.h>
#include <thundersvm/kernel/smo_kernel.h>

real
NuSMOSolver::calculate_rho(const SyncData<real> &f_val, const SyncData<int> &y, SyncData<real> &alpha, real C) const {
    int n_free_p = 0, n_free_n = 0;
    real sum_free_p = 0, sum_free_n = 0;
    real up_value_p = INFINITY, up_value_n = INFINITY;
    real low_value_p = -INFINITY, low_value_n = -INFINITY;
    for (int i = 0; i < alpha.size(); ++i) {
        if (y[i] > 0) {
            if (alpha[i] > 0 && alpha[i] < C) {
                n_free_p++;
                sum_free_p += f_val[i];
            }
            if (is_I_up(alpha[i], y[i], C)) up_value_p = min(up_value_p, -f_val[i]);
            if (is_I_low(alpha[i], y[i], C)) low_value_p = max(low_value_p, -f_val[i]);
        } else {
            if (alpha[i] > 0 && alpha[i] < C) {
                n_free_n++;
                sum_free_n += -f_val[i];
            }
            if (is_I_up(alpha[i], y[i], C)) up_value_n = min(up_value_n, -f_val[i]);
            if (is_I_low(alpha[i], y[i], C)) low_value_n = max(low_value_n, -f_val[i]);
        }
    }
    real r1 = n_free_p != 0 ? sum_free_p / n_free_p : (-(up_value_p + low_value_p) / 2);
    real r2 = n_free_n != 0 ? sum_free_n / n_free_n : (-(up_value_n + low_value_n) / 2);
    real r = (r1 + r2) / 2;
    for (int i = 0; i < alpha.size(); ++i) {
        alpha[i] /= r;//TODO parallel
    }
    return (r1 - r2) / 2 / r;
}

void NuSMOSolver::smo_kernel(const int *label, real *f_values, real *alpha, real *alpha_diff, const int *working_set,
                             int ws_size, float C, const float *k_mat_rows, const float *k_mat_diag, int row_len,
                             real eps, real *diff_and_bias) const {
    size_t smem_size = ws_size * sizeof(real) * 3 + 2 * sizeof(float);
    nu_smo_solve_kernel << < 1, ws_size, smem_size >> >
                                         (label, f_values, alpha, alpha_diff,
                                                 working_set, ws_size, C, k_mat_rows,
                                                 k_mat_diag, row_len, eps, diff_and_bias);
}

void NuSMOSolver::select_working_set(vector<int> &ws_indicator, const SyncData<int> &f_idx2sort, const SyncData<int> &y,
                                     const SyncData<real> &alpha, real C, SyncData<int> &working_set) const {
    int n_instances = ws_indicator.size();
    int p_left_p = 0;
    int p_left_n = 0;
    int p_right_p = n_instances - 1;
    int p_right_n = n_instances - 1;
    int n_selected = 0;
    const int *index = f_idx2sort.host_data();
    while (n_selected < working_set.size()) {
        int i;
        if (p_left_p < n_instances) {
            i = index[p_left_p];
            while (ws_indicator[i] == 1 || !(y[i] > 0 && is_I_up(alpha[i], y[i], C))) {
                //construct working set of I_up
                p_left_p++;
                if (p_left_p == n_instances) break;
                i = index[p_left_p];
            }
            if (p_left_p < n_instances) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }
        if (p_left_n < n_instances) {
            i = index[p_left_n];
            while (ws_indicator[i] == 1 || !(y[i] < 0 && is_I_up(alpha[i], y[i], C))) {
                //construct working set of I_up
                p_left_n++;
                if (p_left_n == n_instances) break;
                i = index[p_left_n];
            }
            if (p_left_n < n_instances) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }
        if (p_right_p >= 0) {
            i = index[p_right_p];
            while (ws_indicator[i] == 1 || !(y[i] > 0 && is_I_low(alpha[i], y[i], C))) {
                //construct working set of I_low
                p_right_p--;
                if (p_right_p == -1) break;
                i = index[p_right_p];
            }
            if (p_right_p >= 0) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }
        if (p_right_n >= 0) {
            i = index[p_right_n];
            while (ws_indicator[i] == 1 || !(y[i] < 0 && is_I_low(alpha[i], y[i], C))) {
                //construct working set of I_low
                p_right_n--;
                if (p_right_n == -1) break;
                i = index[p_right_n];
            }
            if (p_right_n >= 0) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }
    }
}
