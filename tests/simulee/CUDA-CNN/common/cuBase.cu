#include "hip/hip_runtime.h"
#include "cuBase.h"

__device__ float d_nonLinearity(float val, int NONLIN){
	if(NONLIN == NL_RELU){
		if(val < 0.0) return 0.0;
		else return val;
	}else if(NONLIN == NL_LRELU){
        if(val < 0.0) return 0.1f * val;
        else return val;
    }else if(NONLIN == NL_TANH){
		return tanh(val * 2.0 / 3.0) * 1.7159;
	}
	else{
		return val;
	}
}

__device__ float d_dnonLinearity(float val,int NONLIN){
	if(NONLIN == NL_RELU){
		if(val > 0.0) return 1.0;
		else return 0.0;
	}else if (NONLIN == NL_LRELU){
        if(val > 0.0) return 1.0;
        else return 0.1;
    }
	else if(NONLIN == NL_TANH){
		float res = 1.7159;
		float temp = val * val / 1.7159;
		res = (res - temp) * 2.0 / 3.0;
		return res;
	}else {
		return val;
	}
}

__global__ void g_dnonLinearity(float* delta, float*acti, int len, int NONLIN)
{
	int skip = gridDim.x * blockDim.x;
	for(int i = 0; i < len; i += skip)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x + i;
		if(id < len)
		{	
			delta[id] *= d_dnonLinearity(acti[id], NONLIN);
		}
	}
}

__global__ void g_nonLinearity(float* inputs, int len, int NONLIN)
{
	for(int i = 0; i < len; i += gridDim.x * blockDim.x)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x + i;
		if(id < len)
		{	
			inputs[id] = d_nonLinearity(inputs[id], NONLIN);
		}
	}
}

__device__ double atomicAdd(double* address, double val)
{ 	
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
			__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__device__ void swap(float& val1, float& val2){
	float tmp = val1;
	val1 = val2;
	val2 = tmp;
}


__global__ void g_vecAdd(float**_v_w, float** _wgrad,float** _w,
	float** _v_b, float** _bgrad, float** _b, 
	int lenw, int lenb,
	float momentum, float lratew, float lrateb)
{
	float* v_w   = _v_w[blockIdx.x];
	float* wgrad = _wgrad[blockIdx.x];
	float* w     = _w[blockIdx.x];
	float* v_b   = _v_b[blockIdx.x];
	float* bgrad = _bgrad[blockIdx.x];
	float* b     = _b[blockIdx.x];

	int idx = threadIdx.x;
	for(int i = 0; i < lenw; i += blockDim.x)
	{
		int id = i + idx;
		if(id < lenw)
		{
			v_w[id] = v_w[id] * momentum + wgrad[id] * lratew;
			w[id] -= v_w[id];
		}
	}
	for(int i = 0; i < lenb; i += blockDim.x)
	{
		int id = i + idx;
		if(id < lenb)
		{
			v_b[id] = v_b[id] * momentum + bgrad[id] * lrateb;
			b[id] -= v_b[id];
		}
	}
}

__global__ void g_vecAdd(float*v_w, float*wgrad,float* w,
	float* v_b, float* bgrad, float* b, 
	int lenw, int lenb,
	float momentum, float lratew, float lrateb)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = 0; i < lenw; i += blockDim.x * gridDim.x)
	{
		int id = i + idx;
		if(id < lenw)
		{
			v_w[id] = v_w[id] * momentum + wgrad[id] * lratew;
			w[id] -= v_w[id];
		}
	}
	for(int i = 0; i < lenb; i += blockDim.x * gridDim.x)
	{
		int id = i + idx;
		if(id < lenb)
		{
			v_b[id] = v_b[id] * momentum + bgrad[id] * lrateb;
			b[id] -= v_b[id];
		}
	}
}


__global__ void g_getCost_3(float* cost,
	float** weight,
	float lambda, int wlen)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = 0;
	__syncthreads();
	float* w = weight[blockIdx.x];

	for(int i = 0; i < wlen; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < wlen)
		{
			_sum[threadIdx.x] += w[id] * w[id];
		}
	}

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}

	if(threadIdx.x == 0)
	{
		atomicAdd(cost, _sum[0] * lambda * 0.5);
	}
}


/*
*/
__global__ void g_getBgrad(float* softMaxDelta, float* bgrad, float* dropb, int batch)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = softMaxDelta[threadIdx.x * gridDim.x + blockIdx.x];

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	if(threadIdx.x == 0)
	{
		bgrad[blockIdx.x] = _sum[0] / batch;
		bgrad[blockIdx.x] *= dropb[blockIdx.x];
	}
}


/*
dim3(curDelta->cols), dim3(curDelta->rows), 
sizeof(float) * curDelta->rows
*/
__global__ void g_getBgrad(float* softMaxDelta, float* bgrad, int batch)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = softMaxDelta[threadIdx.x * gridDim.x + blockIdx.x];

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[blockIdx.x] = _sum[0] / batch;
	}
}

/*
* function: getcost
*/
__global__ void g_getCost_1(float* softMaxP,
	float* groundTruth, float* cost, int*y, int rows, int cols, int batch)
{
	extern __shared__ float _sum[];
	int len = rows * cols;
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			groundTruth[id] = 0;
		}
	}
	__syncthreads();
	for(int i = 0; i < rows; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < rows)
		{
			int yy = y[id];
			groundTruth[id * cols + yy] = 1;
		}
	}
	_sum[threadIdx.x] = 0;
	__syncthreads();
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			_sum[threadIdx.x] += __logf(softMaxP[id] + 1.0e-10) * groundTruth[id];
		}
	}
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		cost[0] = -_sum[0] / batch;
	}
}


__global__ void g_getCost_2(float* cost,
	float* weight,
	float lambda, int len)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = 0;
	__syncthreads();
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			_sum[threadIdx.x] += 0.5 * weight[id] * weight[id];
		}
	}
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		cost[0] += _sum[0] * lambda;
	}
}


/*
function: g_preDeltaFormat
threads : <<<dim3(batch), dim3(512)>>> 
*/
__global__ void g_preDeltaFormat(float* cuPoolFlDelta, 
	float* cuPoolDelta, int batch, int size, int channels){
	int b = blockIdx.x;
	int len = size * channels;
	for(int i = 0; i < len; i += blockDim.x){
		int id = i + threadIdx.x;
		if(id < len){
			int s = id / channels;
			int c = id % channels;
			cuPoolDelta[c * batch * size + b * size + s] = cuPoolFlDelta[b * size * channels + size * c + s];
		}
	}
}


/*
* function: cuMatrix(batch, size, channel) to cuMatrix(batch, size * channel, 1)
* blocks  : dim3(batch)
* threads : dim3(min(512, cuPool[poolidx]->cols))
*/
__global__ void g_convert(float* cuPool, float*cuPoolToFlActi, int batch, int size, int channel){
	int b   = blockIdx.x;
	int len = size * channel;
	for(int i = 0; i < len; i+=blockDim.x){
		int id = i + threadIdx.x;
		if(id < len){
			int s = id / channel;
			int c = id % channel;
			cuPoolToFlActi[b * size * channel + size * c + s] = cuPool[c * batch * size + b * size + s];
		}
	}
}
