#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "atax.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0


#ifndef M_PI
#define M_PI 3.14159
#endif

#define RUN_ON_CPU











__global__ void atax_kernel1(int nx, int ny, DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_NX)
	{
		tmp[i] = 0;
		int j;
		for(j=0; j < _PB_NY; j++)
		{
			tmp[i] += A[i*NY+j] * x[j];
		}
	}
}

__global__ void atax_kernel2(int nx, int ny, DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < _PB_NY)
	{
		y[j] = 0;
		int i;
		for(i=0; i < _PB_NX; i++)
		{
			y[j] += A[i*NY+j] * tmp[i];
		}
	}
}








/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"
