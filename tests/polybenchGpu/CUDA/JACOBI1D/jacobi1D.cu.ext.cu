#include "hip/hip_runtime.h"
/**
 * jacobi1D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>

#define POLYBENCH_TIME 1

#include "jacobi1D.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU








__global__ void runJacobiCUDA_kernel1(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i > 0) && (i < (_PB_N-1)))
	{
		B[i] = 0.33333 * (A[i-1] + A[i] + A[i + 1]);
	}
}


__global__ void runJacobiCUDA_kernel2(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((j > 0) && (j < (_PB_N-1)))
	{
		A[j] = B[j];
	}
}








/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"

