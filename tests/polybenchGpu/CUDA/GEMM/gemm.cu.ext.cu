#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gemm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

#define RUN_ON_CPU














__global__ void gemm_kernel(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NJ))
	{	
		c[i * NJ + j] *= beta;
		int k;
		for(k=0; k < _PB_NK; k++)
		{
			c[i * NJ + j] += alpha * a[i * NK + k] * b[k * NJ +j];
		}
	}
}





/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"
