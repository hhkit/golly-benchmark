#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "2mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU











__global__ void mm2_kernel1(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *tmp, DATA_TYPE *A, DATA_TYPE *B)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NJ))
	{ 
		tmp[i * NJ + j] = 0;
		int k;
		for (k = 0; k < _PB_NK; k++)
		{
			tmp[i * NJ + j] += alpha * A[i * NK + k] * B[k * NJ + j];
		}
	}
}


__global__ void mm2_kernel2(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *tmp, DATA_TYPE *C, DATA_TYPE *D)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NL))
	{ 
		D[i * NL + j] *= beta;
		int k;
		for (k = 0; k < _PB_NJ; k++)
		{
			D[i * NL + j] += tmp[i * NJ + k] * C[k * NL + j];
		}
	}
}





/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */








#include "../../common/polybench.c"
