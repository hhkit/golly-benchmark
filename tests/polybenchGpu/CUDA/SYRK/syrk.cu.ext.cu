#include "hip/hip_runtime.h"
/**
 * syrk.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "syrk.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0


#define RUN_ON_CPU














__global__ void syrk_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *c)
{
	/*  C := alpha*A*A' + beta*C */
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NI))
	{
		c[i * NI + j] *= beta;
		int k;		
		for(k=0; k < _PB_NJ; k++)
		{
			c[i * NI + j] += alpha * a[i * NJ + k] * a[j * NJ + k];
		}
	}
}





/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"

