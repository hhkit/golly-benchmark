#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "fdtd2d.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

#define RUN_ON_CPU















__global__ void fdtd_step1_kernel(int nx, int ny, DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NX) && (j < _PB_NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}



__global__ void fdtd_step2_kernel(int nx, int ny, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < _PB_NX) && (j < _PB_NY) && (j > 0))
	{
		ex[i * NY + j] = ex[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(int nx, int ny, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < (_PB_NX-1)) && (j < (_PB_NY-1)))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * NY + (j+1)] - ex[i * NY + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}





/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"

