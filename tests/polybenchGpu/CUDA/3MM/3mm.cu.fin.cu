#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "3mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU










	
__global__ void mm3_kernel1(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NJ))
	{
		E[i * NJ + j] = 0;
		int k;
		for(k=0; k < _PB_NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}

	
__global__ void mm3_kernel2(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NJ) && (j < _PB_NL))
	{
		F[i * NL + j] = 0;
		int k;
		for(k=0; k < _PB_NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NL))
	{
		G[i * NL + j] = 0;
		int k;
		for(k=0; k < _PB_NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
		}
	}
}


/* Main computational kernel on CPU */






/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"

