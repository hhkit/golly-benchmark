#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "syr2k.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0


#define RUN_ON_CPU














__global__ void syr2k_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NI))
	{
		c[i * NI + j] *= beta;
		
		int k;
		for(k = 0; k < NJ; k++)
		{
			c[i * NI + j] += alpha * a[i * NJ + k] * b[j * NJ + k] + alpha * b[i * NJ + k] * a[j * NJ + k];
		}
	}
}





/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */





#include "../../common/polybench.c"
